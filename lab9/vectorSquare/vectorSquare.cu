/**
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 * Extended for use in CS 374 at Calvin College by Joel C. Adams.
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <omp.h>
#include <math.h> // sqrt

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. 
 * The 3 vectors have the same number of elements numElements.
 */
__global__
void vectorSquare(const float *A, float *C, unsigned long numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] * A[i];
    }
}

void checkErr(hipError_t err, const char* msg) 
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s (error code %d: '%s')!\n", msg, err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Host main routine
 */
int main(int argc, char** argv)
{
    
    double totalCuda,        // total cuda compute time
            hostDev,         // host -> device array copy time 
            arrayCompute,    // time spent computing the array
            devHost,         // device -> host array copy time
            totalSequential; // sequential time
    
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    unsigned long numElements = 50000;
    if (argc == 2) {
      numElements = strtoul( argv[1] , 0, 10 );
    }
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %lu elements]\n", numElements);

    // Allocate the host input vectors A & B
    float * h_A = (float *)malloc(size);

    // Allocate the host output vector C
    float * h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
    }

    // 1a. Allocate the device input vectors A & B
    float * d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    checkErr(err, "Failed to allocate device vector A");

    // 1.b. Allocate the device output vector C
    float * d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    checkErr(err, "Failed to allocate device vector C");

    totalCuda = omp_get_wtime();

    hostDev = omp_get_wtime();
    
    // 2. Copy the host input vectors A and B in host memory 
    //     to the device input vectors in device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device vector A from host to device");

    hostDev = omp_get_wtime() - hostDev;

    arrayCompute = omp_get_wtime();

    // 3. Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorSquare<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, numElements);
    err = hipGetLastError();
    checkErr(err, "Failed to launch vectorAdd kernel");

    arrayCompute = omp_get_wtime() - arrayCompute;

    devHost = omp_get_wtime();

    // 4. Copy the device result vector in device memory
    //     to the host result vector in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    checkErr(err, "Failed to copy vector C from device to host");
    
    devHost = omp_get_wtime() - devHost;

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (!(fabs(h_A[i] * h_A[i]) < INT_MAX))
        {
            fprintf(stderr, "Result verification %lf failed at element %d!\n", h_A[i], i);
            exit(EXIT_FAILURE);
        }
    }
    
    totalCuda = omp_get_wtime() - totalCuda;
    
    printf("CUDA test PASSED\n");
    printf("CUDA time: %lf\nHost -> device array cpy: %lf\nArray compute: %lf\nDevice -> host cpy time: %lf\n",
        totalCuda, hostDev, arrayCompute, devHost); 

    // Free device global memory
    err = hipFree(d_A);
    checkErr(err, "Failed to free device vector A");

    err = hipFree(d_C);
    checkErr(err, "Failed to free device vector C");

    totalSequential = omp_get_wtime();
    
    // repeat the computation sequentially
    for (int i = 0; i < numElements; ++i)
    {
       h_C[i] = sqrt(h_A[i]);
    }

    // verify again
    for (int i = 0; i < numElements; ++i)
    {
        if (!(fabs(h_A[i] * h_A[i]) < INT_MAX))
        {
            fprintf(stderr, "Result verification %lf failed at element %d!\n", h_A[i], i);
            exit(EXIT_FAILURE);
        }
    }
    
    totalSequential = omp_get_wtime() - totalSequential;
    
    printf("\nNormal test PASSED\n");
    printf("Normal time: %lf\n", totalSequential); 
    
    // Free host memory
    free(h_A);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();
    checkErr(err, "Unable to reset device");

    printf("Done\n");
    return 0;
}
