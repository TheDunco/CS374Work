/**
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 * Extended for use in CS 374 at Calvin College by Joel C. Adams.
 * Modified by Duncan Van Keulen for Homework 9 for HPC at Calvin University
 * 22 November 2021
 * 
 * Vector hypotenuse: C = sqrt(A * A + B * B)
 */

#include <stdio.h>
#include <omp.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. 
 * The 3 vectors have the same number of elements numElements.
 */
__global__
void vectorHypot(const float *A, const float *B, float *C, unsigned long numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = sqrt(A[i]*A[i] + B[i]*B[i]);
    }
}

void checkErr(hipError_t err, const char* msg) 
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s (error code %d: '%s')!\n", msg, err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Host main routine
 */
int main(int argc, char** argv)
{
    
    double totalCuda,        // total cuda compute time
            hostDev,         // host -> device array copy time 
            arrayCompute,    // time spent computing the array
            devHost,         // device -> host array copy time
            totalSequential; // sequential time
    
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    unsigned long numElements = 50000;
    if (argc == 2) {
      numElements = strtoul( argv[1] , 0, 10 );
    }
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %lu elements]\n", numElements);

    // Allocate the host input vectors A & B
    float * h_A = (float *)malloc(size);
    float * h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float * h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // 1a. Allocate the device input vectors A & B
    float * d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    checkErr(err, "Failed to allocate device vector A");
    float * d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    checkErr(err, "Failed to allocate device vector B");

    // 1.b. Allocate the device output vector C
    float * d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    checkErr(err, "Failed to allocate device vector C");

    totalCuda = omp_get_wtime();

    hostDev = omp_get_wtime();
    
    // 2. Copy the host input vectors A and B in host memory 
    //     to the device input vectors in device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device vector A from host to device");


    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device vector B from host to device");
    
    hostDev = omp_get_wtime() - hostDev;

    arrayCompute = omp_get_wtime();

    // 3. Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorHypot<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();
    checkErr(err, "Failed to launch vectorAdd kernel");

    arrayCompute = omp_get_wtime() - arrayCompute;

    devHost = omp_get_wtime();

    // 4. Copy the device result vector in device memory
    //     to the host result vector in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    checkErr(err, "Failed to copy vector C from device to host");
    
    devHost = omp_get_wtime() - devHost;

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_C[i]) != fabs(sqrt(h_A[i]*h_A[i] + h_B[i]*h_B[i])))
        {
            fprintf(stderr, "Result verification failed at element %d. %lf!\n", i, h_C[i]);
            exit(EXIT_FAILURE);
        }
    }
    
    totalCuda = omp_get_wtime() - totalCuda;
    
    printf("CUDA test PASSED\n");
    printf("CUDA time: %lf\nHost -> device array cpy: %lf\nArray compute: %lf\nDevice -> host cpy time: %lf\n",
        totalCuda, hostDev, arrayCompute, devHost); 

    // Free device global memory
    err = hipFree(d_A);
    checkErr(err, "Failed to free device vector A");

    err = hipFree(d_B);
    checkErr(err, "Failed to free device vector B");

    err = hipFree(d_C);
    checkErr(err, "Failed to free device vector C");

    totalSequential = omp_get_wtime();
    
    // repeat the computation sequentially
    for (int i = 0; i < numElements; ++i)
    {
       h_C[i] = h_A[i] + h_B[i];
    }

    // verify again
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_C[i]) != fabs(sqrt(h_A[i]*h_A[i] + h_B[i]*h_B[i])))
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    
    totalSequential = omp_get_wtime() - totalSequential;
    
    printf("\nNormal test PASSED\n");
    printf("Normal time: %lf\n", totalSequential); 
    
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();
    checkErr(err, "Unable to reset device");

    printf("Done\n");
    return 0;
}

